#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<stddef.h>
#include<sys/time.h>
#define BILLION  1000L;

__global__ void findText(char *dbuffer,char* dword,int *dcount,int overlap,int blockSize,int noOfThreads){
int i = blockDim.x * blockIdx.x + threadIdx.x;
char *temp;
int ix;
char *s1,*s2;
int length,cvalue=0;
char *temp1,*temp2;int tempi;
if(i < noOfThreads){
//printf("\ni=%d Looking for %s in %d blocks-->",i,word,noOfThreads);
//printf("");
	temp = dbuffer+(blockSize*i);
//printf("\nIncrementing by : %d",blockSize*i);
	length=blockSize+overlap;
	for (ix = 0; dword[ix] != '\0'; ix++) ;
	//printf("\nwordlength : %d\n",length);
	while(temp!=NULL){
		s1=temp;s2=dword;
		


while(length > 0){

        if(*s1!=*s2)
                s1++;
        else
        {
	        temp1=++s1;temp2=s2+1;tempi=ix-1;
        	while(tempi>0){
	                if(*temp1==*temp2)
        	        {temp2++;temp1++;}
                	else
               		{
                	        break;
               		}
	                tempi--;
       		}
 	       if(tempi==0){
       		        temp = s1;

			break;
       		}
        }
	length--;
}
if(length==0)
temp =  NULL;


		if(temp != NULL){
			cvalue++;
			dcount[i]=dcount[i]+1;
		}
	}

	//dcount[i] = cvalue;
//	__syncthreads();

}
}

int main(int argc, char **argv){

//Initialize things to pass
char *buffer,*dbuffer;
char *word,*dword;
int overlap;
int blockSize;
int noOfThreads ;
int *count,*dcount,totalcount = 0;
long fSize;
int i,threadsPerBlock,blocksPerGrid,var;
FILE *pFile = stdin; 
struct timespec start, stop; 
long accum;

	word = argv[1];


	/*  Initialize Buffer */
	fseek(pFile , 0, SEEK_END);
	fSize = ftell(pFile);
	rewind (pFile);
	buffer = (char*) malloc (sizeof(char)*fSize);
	if (buffer== NULL) {fputs (" \n My Memory error",stderr); exit (2);}
	if(fread(buffer,fSize*sizeof(char),1,pFile)!=1){fputs(" \n My Memory Err",stderr); exit(2);}


	/*Initialize grid numbers*/ 
	blocksPerGrid = 1;

for (var =1 ; var < 512; var++){
	totalcount=0;
	threadsPerBlock = var;
	noOfThreads = threadsPerBlock * blocksPerGrid;
	blockSize=(long)fSize/noOfThreads;
	//printf("\nnoOfThreads  = %d blockSize = %ld\n fileSize = %ld",noOfThreads,blockSize,fSize);

	/*Initialize count*/ 
	count = (int*) malloc (sizeof(int) * noOfThreads);
	//hdbgarr = (int*) malloc (sizeof(int) * noOfThreads);
	
	/*Initialize word*/ 
	hipMalloc((void**)&dword,sizeof(char)*(strlen(word)));
	hipMalloc((void**)&dbuffer,fSize*sizeof(char));
	hipMalloc((void**)&dcount,sizeof(int)*noOfThreads);

	/*  Initialize overlap which is one less than strlen */
	overlap = strlen(word)-1;


	for(i=0;i<noOfThreads;i++) count[i]=0;
	hipMemcpy(dword,word,sizeof(char)*strlen(word),hipMemcpyHostToDevice);
	hipMemcpy(dbuffer,buffer,fSize*sizeof(char),hipMemcpyHostToDevice);
	/*Copy count*/ 
	hipMemcpy(dcount,count,(sizeof(int)*noOfThreads),hipMemcpyHostToDevice);


	//printf(" \n \n All Initialized");

if( clock_gettime( CLOCK_REALTIME, &start) == -1 ) {
      perror( "clock gettime" );
      return EXIT_FAILURE;
}


	findText<<<blocksPerGrid,threadsPerBlock>>>(dbuffer,dword,dcount,overlap,blockSize,noOfThreads);//passing noofthredas-1 as padding nt handles yet

	hipDeviceSynchronize();

if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) {
      perror( "clock gettime" );
      return EXIT_FAILURE;
    }
	hipMemcpy(count,dcount,(sizeof(int)*noOfThreads),hipMemcpyDeviceToHost);
	

//printf("%s\n\n",buffer);
	//printf("\nCounts");
	for(i=0;i<noOfThreads;i++){
		//printf("%d ",count[i]);
		totalcount += count[i];
	}
	//printf(" \n Total Count = %d",totalcount);

	accum = (stop.tv_nsec - start.tv_nsec)/BILLION;
	printf("\n %d, %ld",var,accum);

	//free(word);
	free(count);
	hipFree(dword);
	hipFree(dbuffer);
	hipFree(dcount);
}
	printf(" \n Total Count = %d",totalcount);
	free(buffer);
}
